#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <math.h>
#include <time.h>
#include "init_heat_source.c"

#define GIG 1000000000
#define CPG 2.0
// Assertion to check for errors
#define CUDA_SAFE_CALL(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
	if (code != hipSuccess) 
	{
		fprintf(stderr,"CUDA_SAFE_CALL: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

#define NUM_THREADS_PER_BLOCK 	256
#define NUM_BLOCKS 				1024
#define PRINT_TIME 				0
#define SM_ARR_LEN				512
#define TOL						1

#define V_PRINT					0
#define CPU_VALIDATE			0

#define RHO 					0.1
#define ITERS 					1000

#define DRIFT					0.4

#define HEATER_TEMP				85
#define KHAI_TEMP 				98
#define OUTSIDE_TEMP			20
#define WALL_TEMP 				60
#define ROOM_TEMP				50


#define IMUL(a, b) __mul24(a, b)

void initializeArray1D(float *arr, int len, int size, float window1, float window2);

// y is starting point, result is finishing
// each kernel iteration is actually 2 relaxation calls
__global__ void kernel_sor_2d (int arrLen, int arrSize, float* x, float* y, float* result, float w_drift) {
	const int tid = IMUL(blockDim.x, blockIdx.x) + threadIdx.x;
	const int threadN = IMUL(blockDim.x, gridDim.x);
	
	__shared__ float mw_drift;
	mw_drift = ((1.0-w_drift)/3.0);

	int i;
	__shared__ int ignoreFlag[256];

	/* Walls & Heater */
	for(i = 0; i < 256; i++)
	if ((blockIdx.y == 0 || blockIdx.y == 31) || (blockIdx.x == 0 || blockIdx.x == 31) ||
		((blockIdx.x >= 28 && blockIdx.x < 31) && (blockIdx.y >= 2 && blockIdx.y < 11)))
		ignoreFlag[i] = 1;
	else
		ignoreFlag[i] = 0;

	/* Relax Here, Get Result from y, Get y from Result */
	for(i = tid; i < arrSize; i += threadN) {
		
		__syncthreads();
		if (ignoreFlag[tid%256]) {
			result[i] = RHO * y[i] + (1.0-RHO) * (
					w_drift *(y[((((i+arrLen)%arrSize))+arrSize)%arrSize]) +	//top    
					mw_drift*((y[(((i+1)%arrSize)+arrSize)%arrSize]) + 			//right  
							(y[(((i-1)%arrSize)+arrSize)%arrSize]) + 			//left
							(y[((((i-arrLen)%arrSize))+arrSize)%arrSize]))		//bottom
			);
		}
		else {
			result[i] = y[i];
		}

		__syncthreads();
		if (ignoreFlag[tid%256]) {
			y[i] = RHO * result[i] + (1.0-RHO) * (
					w_drift *(result[((((i+arrLen)%arrSize))+arrSize)%arrSize]) +	//top  	 
					mw_drift*((result[(((i+1)%arrSize)+arrSize)%arrSize]) + 		//right  
							(result[(((i-1)%arrSize)+arrSize)%arrSize]) + 			//left
							(result[((((i-arrLen)%arrSize))+arrSize)%arrSize])) 	//bottom
			);
		}
		else 
		{
			y[i] = result[i];	
		}

		__syncthreads();
		x[i] = y[i] - result[i];	// get residual array
	}
	__syncthreads();
}

int main(int argc, char **argv){
	int arrLen = 0;
	int arrSize = 0;
	int iterCount = 0;
	float w_drift;

	// GPU Timing variables
	hipEvent_t start, stop;
	float elapsed_gpu;
	float total_time = 0;
	
	// Arrays on GPU global memoryc
	float *d_x;
	float *d_y;
	float *d_result;

	// Arrays on the host memory
	float *h_x;
	float *h_y;
	float *h_result;
	float *h_result_gold;

	// CPU Timers
	timespec time1, time2, elapsed_cpu;
	timespec diff(struct timespec start, struct timespec end);
	
	int i, errCount = 0, zeroCount = 0;
	float window1, window2;
	
	if (argc > 1) {
		arrLen  = atoi(argv[1]);
	}
	else {
		arrLen = SM_ARR_LEN;
	}

	arrSize = arrLen*arrLen;

	//printf("Length of the array = %d\n", arrLen);

    // Select GPU
    CUDA_SAFE_CALL(hipSetDevice(0));

	// Allocate GPU memory
	size_t allocSize = arrSize * sizeof(float);
	CUDA_SAFE_CALL(hipMalloc((void **)&d_x, allocSize));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_y, allocSize));
	CUDA_SAFE_CALL(hipMalloc((void **)&d_result, allocSize));
		
	// Allocate arrays on host memory
	h_x                        = (float *) malloc(allocSize);	// this is our error
	h_y                        = (float *) malloc(allocSize);	// this is our result every even
	h_result                   = (float *) malloc(allocSize);
	h_result_gold              = (float *) malloc(allocSize);

	int scale;
	float sumFaceTemp;
	float count;


	window1 = 0.8;
	window2 = 0.5;				
	count = 0.0;
	sumFaceTemp = 0.0;
	w_drift = .25 + 0.25 * (window1 + window2);

	// Initialize the host arrays
	//printf("\nInitializing the arrays ...");
	// Arrays are initialized with a known seed for reproducability
	initializeArray1D(h_x, arrLen, arrSize, window1, window2); //2453
	initializeArray1D(h_y, arrLen, arrSize, window1, window2); //1467
	//printf("\t... done\n\n");

	// int i, j;
	// for(i = 32; i < 128; i++)
	// {
	// 	for(j = 32; j < 64; j++)
	// 	{
	// 		sumFaceTemp += h_y[j*arrLen+i];
	// 		count += 1.0;
	// 		printf("Temperature at i: %d j: %d -- %.15f\n", i, j, h_y[i*arrLen + j]);
	// 	}
	// }
				
				
		#if PRINT_TIME
			// Create the cuda events
			hipEventCreate(&start);
			hipEventCreate(&stop);
			// Record event on the default stream
			hipEventRecord(start, 0);
		#endif
					
				//printf("GPU work starting ...\n");
				// Transfer the arrays to the GPU memory
				//printf("GPU hipMemcpyHostToDevice...\n");
				CUDA_SAFE_CALL(hipMemcpy(d_x, h_x, allocSize, hipMemcpyHostToDevice));
				CUDA_SAFE_CALL(hipMemcpy(d_y, h_y, allocSize, hipMemcpyHostToDevice));
				  
				// Launch the kernel
				//kernel_add<<<NUM_BLOCKS, NUM_THREADS_PER_BLOCK>>>(arrLen, d_x, d_y, d_result);
				//printf("GPU Kernel running...\n");

				////////////////////////
				// X IS NEVER USED /////
				////////////////////////
				// printf("%f\t%f\t%f\t",window1,window2,w_drift);
				int stopCount = 0;
				int loopFlag = 1;
				do {
					for (iterCount =0; iterCount < ITERS; iterCount++)
					{
						kernel_sor_2d<<<NUM_BLOCKS, NUM_THREADS_PER_BLOCK>>>(arrLen, arrSize, d_x, d_y, d_result, w_drift);
						CUDA_SAFE_CALL(hipDeviceSynchronize());
					}		

					// Check for errors during launch
					//printf("GPU hipPeekAtLastError\n");
					CUDA_SAFE_CALL(hipPeekAtLastError());
					
					// Transfer the results back to the host
					//printf("GPU hipMemcpyDeviceToHost\n");
					CUDA_SAFE_CALL(hipMemcpy(h_y, d_y, allocSize, hipMemcpyDeviceToHost));
					CUDA_SAFE_CALL(hipMemcpy(h_x, d_x, allocSize, hipMemcpyDeviceToHost));
					CUDA_SAFE_CALL(hipDeviceSynchronize());

					stopCount++;
					if(stopCount == 3 || stopCount == 6 || stopCount == 12 || 
						stopCount == 25 || stopCount == 50 || stopCount == 100) {
						printf("scale=%d\n", stopCount);
						int cnti, cntj;
						for(cnti = 0; cnti < 512; cnti++) {
							for(cntj = 0; cntj < 512; cntj++) {
								printf("%d\t%d\t%f\n", cntj, cnti, h_y[cnti*arrLen + cntj]);
							}
						} 
					}
					
					if(stopCount == 100){
						loopFlag = 0;
					}

				}while(loopFlag);

		#if PRINT_TIME
			// Stop and destroy the timer
			hipEventRecord(stop,0);
			hipEventSynchronize(stop);
			hipEventElapsedTime(&elapsed_gpu, start, stop);
			total_time = total_time + elapsed_gpu;
			//printf("\nGPU time: %f (msec)\n", elapsed_gpu);
			hipEventDestroy(start);
			hipEventDestroy(stop);
		#endif
			/******************
					ADD
					THIS
					BACK
					IN 
					LATER
					PLS

			******************/
				// if ((sumFaceTemp / count) > 68.00 && (sumFaceTemp / count) < 70.00) {
				// 	break;
				// }

#if PRINT_TIME
	printf("\n\nTotal Time: %f (msec)\n", total_time);
#endif
	

	
#if CPU_VALIDATE
	// Compute the results on the host
	/**/
	clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &time1);
	for(iterCount = 0; iterCount < ITERS/2; iterCount++){
		for(i = 0; i < arrSize; i++) {
			h_result_gold[i] = RHO * h_y[i]
				+ (0.25) * (1-RHO) * (	
					h_y[(((i+1)%arrSize)+arrSize)%arrSize] + h_y[((((i+arrLen)%arrSize))+arrSize)%arrSize] +
					h_y[(((i-1)%arrSize)+arrSize)%arrSize] + h_y[((((i-arrLen)%arrSize))+arrSize)%arrSize]
			);
		}

		for(i = 0; i < arrSize; i++) {
			h_y[i] = RHO * h_result_gold[i]
				+ (0.25) * (1-RHO) * (	
					h_result_gold[(((i+1)%arrSize)+arrSize)%arrSize] + h_result_gold[((((i+arrLen)%arrSize))+arrSize)%arrSize] +
					h_result_gold[(((i-1)%arrSize)+arrSize)%arrSize] + h_result_gold[((((i-arrLen)%arrSize))+arrSize)%arrSize]
			);
		}
	}
	clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &time2);

	elapsed_cpu = diff(time1, time2);
	long long int cputime = (long long int)((double)(CPG)*(double)(GIG * elapsed_cpu.tv_sec + elapsed_cpu.tv_nsec));
	printf("\nCPU time: %li (ns)\n", cputime);
	/**/


	/* Print the stuff */

	if (V_PRINT){
		printf("\n");
		for (i = 0; i < arrSize; i++)
		{
			printf("%.6f\t", h_x[i]);
			if (i%arrLen == arrLen-1)
				printf("\n");
		}
		
		printf("\n");
		for (i = 0; i < arrSize; i++)
		{
			printf("%.6f\t", h_result[i]);
			if (i%arrLen == arrLen-1)
				printf("\n");
		}
		/**/
		printf("\n");
		for (i = 0; i < arrSize; i++)
		{
			printf("%.6f\t", h_result_gold[i]);
			if (i%arrLen == arrLen-1)
				printf("\n");
		}
		/**/
	}

	/* --------------- */!

	// Compare the results
	/**/
	for(i = 0; i < arrSize; i++) {
		if (abs(h_result_gold[i] - h_result[i]) > TOL) {
			errCount++;
		}
		if (h_result[i] == 0) {
			zeroCount++;
		}
	}
	/**/

	/*
	for(i = 0; i < 50; i++) {
		printf("%d:\t%.8f\t%.8f\n", i, h_result_gold[i], h_result[i]);
	}
	*/

	/**/
	if ( V_PRINT && errCount > 0) {
		printf("\n@ERROR: TEST FAILED: %d results did not matched\n", errCount);
	}
	else if ( V_PRINT && zeroCount > 0){
		printf("\n@ERROR: TEST FAILED: %d results (from GPU) are zero\n", zeroCount);
	}
	else {
		printf("\nTEST PASSED: All results matched\n");
	}
	/**/
#endif

	//Free-up device and host memory
	CUDA_SAFE_CALL(hipFree(d_x));
	CUDA_SAFE_CALL(hipFree(d_y));
	CUDA_SAFE_CALL(hipFree(d_result));
		   
	free(h_x);
	free(h_y);
	free(h_result);
		
	return 0;
}

void initializeArray1D(float *arr, int len, int size, float window1, float window2) {
	int i;

	for (i = 0; i < size; i++) {
		arr[i] = (float) 80;
	}

	init_heat_source(arr, KHAI_TEMP, OUTSIDE_TEMP, HEATER_TEMP, WALL_TEMP, window1, window2, len);

	//for(i = 0; i < size; i++) {
	//	printf("here i am temperature: %.15f\n", arr[i]);
	//}
}


struct timespec diff(struct timespec start, struct timespec end)
{
  struct timespec temp;
  if ((end.tv_nsec-start.tv_nsec)<0) {
    temp.tv_sec = end.tv_sec-start.tv_sec-1;
    temp.tv_nsec = 1000000000+end.tv_nsec-start.tv_nsec;
  } else {
    temp.tv_sec = end.tv_sec-start.tv_sec;
    temp.tv_nsec = end.tv_nsec-start.tv_nsec;
  }
  return temp;
}